#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "windows.h"
#include <limits>
#include <cmath>
#include <sys\timeb.h>

#include <stdio.h>

#define BLOCK_SIZE 16
#define N 512

hipError_t mulWithCuda(float* c, const float* a, const float* b, size_t size);

__global__ void mulKernel(float* c, const float* a, const float* b)
{
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	float sum = 0;	

	for(int k = 0; k < N; ++k){
        float a_elem = a[i * N + k];
        float b_elem = b[k * N + j];
        sum += a_elem * b_elem;
    }

	c[i * N + j] = sum;
}

int main()
{
    float* a = new float[N * N];
    float* b = new float[N * N];
    float* c_cuda = new float[N * N];
    float* c_omp  = new float[N * N];
    float* c_cpu  = new float[N * N];

    LARGE_INTEGER frequency;        // ticks per second
    LARGE_INTEGER t1, t2;           // ticks
    double elapsedTime;

	for(int i = 0; i < N; ++i) {
		for(int j = 0; j < N; ++j) {
			a[i * N + j] = 1.0f;
			b[i * N + j] = 1.0f;
			c_cuda[i * N + j] = 0.0f;
            c_omp[i * N + j]  = 0.0f;
            c_cpu[i * N + j]  = 0.0f;
		}		
	}

    QueryPerformanceFrequency(&frequency);   

    // Multiply on CPU
    printf("Multiplication on CPU...\n");

    // start timer
    QueryPerformanceCounter(&t1);
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {            
            for(int k = 0; k < N; k++) {
               c_cpu[i * N + j] += a[i * N + k] * b[k * N + j];                 
            }                   
        }
    }
    // stop timer
    QueryPerformanceCounter(&t2);

     // compute and print the elapsed time in millisec
    elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
    printf("done in %f ms\n\n", elapsedTime); 

    // Multiply on CPU with OpenMP
    printf("Multiplication with OMP...\n");

    // start timer
    QueryPerformanceCounter(&t1);
    #pragma omp parallel for num_threads(2)
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            for(int k = 0; k < N; k++) {
                c_omp[i + N * j] += a[i + N * k] * b[k + N * j]; 
            }            
        }
    }
    // stop timer
    QueryPerformanceCounter(&t2);

     // compute and print the elapsed time in millisec
    elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
    printf("done in %f ms\n\n", elapsedTime); 

    //Multiply on GPU
    printf("Multiplication with CUDA...\n");

    QueryPerformanceCounter(&t1);
    // Add vectors in parallel.
    hipError_t cudaStatus = mulWithCuda(c_cuda, a, b, N*N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mulWithCuda failed!");
        return 1;
    }
    // stop timer
    QueryPerformanceCounter(&t2);

     // compute and print the elapsed time in millisec
    elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
    printf("done in %f ms\n\n", elapsedTime); 

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		system("pause");
        return 1;
    }

    int errors = 0;

    for(int i = 0; i < N*N; i++) {
        if(c_cuda[i] != c_cpu[i]) {
            //printf("Error in %d element\n", i);
            errors ++;
        }
    }   

    if(errors == 0) {
        printf("No errors.\n");
    }
    else {
        printf("%d errors\n", errors);
    }  
	
	system("pause");
    return 0;
}

// Helper function for using CUDA to multiply vectors in parallel.
hipError_t mulWithCuda(float* c, const float* a, const float* b, size_t size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;	

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		system("pause");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc(&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        hipError_t error = hipGetLastError();
        fprintf(stderr,"ERROR: hipMalloc: %s\n", hipGetErrorString(error));

		system("pause");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        hipError_t error = hipGetLastError();
        fprintf(stderr,"ERROR: hipMalloc: %s\n", hipGetErrorString(error));

		system("pause");
        goto Error;
    }

    cudaStatus = hipMalloc(&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        hipError_t error = hipGetLastError();
        fprintf(stderr,"ERROR: hipMalloc: %s\n", hipGetErrorString(error));

		system("pause");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		system("pause");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		system("pause");
        goto Error;
    }
	
	
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE); // block size -- threads per block - X*X
    dim3 numBlocks(N/threadsPerBlock.x, 
                   N/threadsPerBlock.y); //grid size -- number of blocks


    // Launch a kernel on the GPU with one thread for each element.
    mulKernel<<<numBlocks, threadsPerBlock>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		system("pause");
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		system("pause");
        goto Error;
    }    

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);	
    
    return cudaStatus;
}
