/* 
 * Compile with
 * nvcc -arch=sm_11 bitonic_sort.cu
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 512 // 2^9
#define BLOCKS 128 // 2^15
#define NUM_VALS THREADS*BLOCKS

double getElapsedTime(clock_t start, clock_t stop)
{
    return ((double) (stop - start)) / CLOCKS_PER_SEC;
    //printf("Elapsed time: %.3fs\n", elapsed);
}

void fillArray(float *array, int length)
{
    srand(time(NULL));
    int i;
    for (i = 0; i < length; i++) {
        array[i] = (float)rand()/(float)RAND_MAX;
    }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        if ((i&k)==0) {
            /* Sort ascending */
            if (dev_values[i]>dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i&k)!=0) {
            /* Sort descending */
            if (dev_values[i]<dev_values[ixj]) {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void gpuBitonicSort(float *values)
{
    float *devValues;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void**) &devValues, size);
    hipMemcpy(devValues, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= NUM_VALS; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
            bitonic_sort_step<<<blocks, threads>>>(devValues, j, k);
        }
    }

    hipDeviceSynchronize();

    hipMemcpy(values, devValues, size, hipMemcpyDeviceToHost);
    hipFree(devValues);
}

const int ASCENDING = 1;
const int DESCENDING = 0;

void compare(float *a, int i, int j, int dir)
{
  if (dir == (a[i] > a[j])) {
        float h=a[i];
        a[i]=a[j];
        a[j]=h;
    }
}

void cpuBitonicMerge(float* values, int lo, int count, int direction)
{
    if (count > 1) {
        int k = count/2;
      
        for (int i=lo; i<lo+k; i++)
           compare(values, i, i+k, direction);
        cpuBitonicMerge(values, lo, k, direction);
        cpuBitonicMerge(values, lo+k, k, direction);
    }
}

void cpuBitonicSort(float *values, int lo, int count, int direction)
{
    if (count > 1) {
        int k = count/2;
        cpuBitonicSort(values, lo, k, ASCENDING);
        cpuBitonicSort(values, lo + k, k, DESCENDING);
        cpuBitonicMerge(values, lo, count, direction);
    }
}

int floatcomp(const void* elem1, const void* elem2)
{
    if(*(const float*)elem1 < *(const float*)elem2)
        return -1;
    return *(const float*)elem1 > *(const float*)elem2;
}

int main(void)
{
    clock_t start, stop;

    float *array    = (float*) malloc( NUM_VALS * sizeof(float));
    float *cpuArray = (float*) malloc( NUM_VALS * sizeof(float));
    float *gpuArray = (float*) malloc( NUM_VALS * sizeof(float));
    
    fillArray(array, NUM_VALS);
    for(int i = 0; i < NUM_VALS; i++) {
        cpuArray[i] = array[i];
        gpuArray[i] = array[i];
    }

    start = clock();
    gpuBitonicSort(gpuArray);
    stop = clock();

    printf("\nElapsed time on gpu: %.3fs\n", getElapsedTime(start, stop));

    start = clock();
    cpuBitonicSort(cpuArray, 0, NUM_VALS, ASCENDING);
    stop = clock();

    printf("\nElapsed time on cpu: %.3fs\n", getElapsedTime(start, stop));    
   
    //std::qsort(array, NUM_VALS, sizeof(float), floatcomp);
    for(int i = 0; i < NUM_VALS; i++) {
        if(gpuArray[i] != cpuArray[i]){
            printf("Error [%d] %f != %f\n", i, gpuArray[i], cpuArray[i]);
        }
    } 
  
    free(array);
    free(cpuArray);
    free(gpuArray);
}
